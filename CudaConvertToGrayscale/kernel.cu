#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <string>
#include <time.h>

class Pixel {
private:
    unsigned char r, g, b;

public:
    Pixel(unsigned char red = 0, unsigned char green = 0, unsigned char blue = 0) {
        r = red;
        g = green;
        b = blue;
    }

    __host__ __device__ bool isBlackAndWhite() {
        return r == g && g == b;
    }

    __host__ __device__ bool isBlack() {
        return r == 0 && g == 0 && b == 0;
    }

    __host__ __device__ bool isWhite() {
        return r == 255 && g == 255 && b == 255;
    }

    __host__ __device__ unsigned char getBrightness() {
        unsigned char brightness = ((int)r + (int)g + (int)b) / 3;
        return brightness;
    }

    __host__ __device__ void setBrightness(unsigned char brightness) {
        r = brightness;
        g = brightness;
        b = brightness;
    }
};

// CUDA kernel function to convert the image to grayscale and create a binary image
__global__ void convert_to_grayscale_kernel(Pixel* pixels, long N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        if (pixels[idx].isBlack() || pixels[idx].isWhite() || pixels[idx].isBlackAndWhite()) {
            return;
        }

        unsigned char brightness = pixels[idx].getBrightness();
        pixels[idx].setBrightness(brightness);
    }
}

void cuda_implementation(std::string& load_image)
{
    static int run = 0;
    clock_t start = clock();

    // Load image file
    std::ifstream file(load_image, std::ios::binary);
    if (!file) {
        printf("Could not open image file.\n");
        return;
    }

    std::string format;
    std::getline(file, format);

    int width, height, maxval;
    file >> width >> height >> maxval;
    long N = width * height;
    file.ignore(1);

    // Read pixel data
    Pixel* host_pixels = new Pixel[N];
    file.read((char*)host_pixels, N * sizeof(Pixel));

    // Allocate memory on the device
    Pixel* device_pixels;
    hipMalloc(&device_pixels, N * sizeof(Pixel));

    // Copy data to to device
    hipMemcpy(device_pixels, host_pixels, N * sizeof(Pixel), hipMemcpyHostToDevice);
    
    // Launch the kernel to convert the image to grayscale
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    convert_to_grayscale_kernel << <blocksPerGrid, threadsPerBlock >> > (device_pixels, N);
    hipDeviceSynchronize();

    /*
        // Functioneaza

        // Define the grid structure
        dim3 block_size(256, 1, 1);
        dim3 grid_size(N / block_size.x + 1, 1, 1);
        convert_to_grayscale_kernel << <block_size, grid_size >> > (device_pixels, N);
        hipDeviceSynchronize();
        
    
        // NU functioneaza

        // Define the grid structure
        dim3 block_size(32, 32, 1);
        dim3 grid_size(N / block_size.x + 1, N / block_size.y + 1, 1);
        convert_to_grayscale_kernel << <block_size, grid_size >> > (device_pixels, N);
        hipDeviceSynchronize();
    */
    
    // Copy the results back to the host
    hipMemcpy(host_pixels, device_pixels, N * sizeof(Pixel), hipMemcpyDeviceToHost);

    // Ignore the first call
    if (run == 0) {
        run++;
        return;
    }

    // Save binary image to file
    std::string save_image = load_image;
    save_image.resize(save_image.length() - 4);
    save_image += "_cuda.ppm";
    std::ofstream outFile(save_image, std::ios::binary);
    outFile << format << "\n" << width << " " << height << "\n" << maxval << "\n";
    outFile.write((const char*)host_pixels, N * sizeof(Pixel));

    // Free memory
    hipFree(device_pixels);
    delete[] host_pixels;

    clock_t end = clock();

    printf("Time elapsed: %.5f seconds (CUDA implementation)\n", float(end - start) / CLOCKS_PER_SEC);
}

int main() {

    // Ignore the first call as it causes elapsed time to increase
    std::string ignore = "ignore.ppm";
    cuda_implementation(ignore);

    std::string img256px = "image256px.ppm";
    std::cout << "Smaller images:\n";
    cuda_implementation(img256px);

    std::string img4k = "image4k.ppm";
    std::cout << "\nLarger images:\n";
    cuda_implementation(img4k);
    return 0;
}